#include "hip/hip_runtime.h"
// **************************************************************************
//                                lj_expand.cu
//                             -------------------
//                            Inderaj Bains (NVIDIA)
//
//  Device code for acceleration of the lj/expand pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : 
//    email                : ibains@nvidia.com
// ***************************************************************************/

#ifdef NV_KERNEL
#include "preprocessor.h"
texture<float4> pos_tex;
#ifndef _DOUBLE_DOUBLE
__inline float4 fetch_pos(const int& i, const float4 *pos) 
  { return tex1Dfetch(pos_tex, i); }
#endif
#endif

__kernel void kernel_pair(__global numtyp4 *x_, __global numtyp4 *lj1,
                          __global numtyp4* lj3, const int lj_types, 
                          __global numtyp *sp_lj_in, __global int *dev_nbor, 
                          __global int *dev_packed, __global acctyp4 *ans,
                          __global acctyp *engv, const int eflag,
                          const int vflag, const int inum,
                          const int nbor_pitch, const int t_per_atom) {
  int tid=THREAD_ID_X;
  int ii=mul24((int)BLOCK_ID_X,(int)(BLOCK_SIZE_X)/t_per_atom);
  ii+=tid/t_per_atom;
  int offset=tid%t_per_atom;

  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  
  if (ii<inum) {
    __global int *nbor=dev_nbor+ii;
    int i=*nbor;
    nbor+=nbor_pitch;
    int numj=*nbor;
    nbor+=nbor_pitch;

    int n_stride;
    __global int *list_end;
    if (dev_nbor==dev_packed) {
      list_end=nbor+mul24(numj,nbor_pitch);
      nbor+=mul24(offset,nbor_pitch);
      n_stride=mul24(t_per_atom,nbor_pitch);
    } else {
      nbor=dev_packed+*nbor;
      list_end=nbor+numj;
      n_stride=t_per_atom;
      nbor+=offset;
    }
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<list_end; nbor+=n_stride) {
  
      int j=*nbor;
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;
        
      int mtype=itype*lj_types+jtype;
      if (r2inv<lj1[mtype].z) {
        numtyp r = sqrt(r2inv);
	numtyp rshift = r - lj1[mtype].w;
	numtyp rshiftsq = rshift*rshift;
	r2inv = (numtyp) 1.0/rshiftsq;
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        force*=factor_lj/rshift/r;
      
        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          energy+=factor_lj*(e-lj3[mtype].z); 
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  
  // Reduce answers
  if (t_per_atom>1) {
    __local acctyp red_acc[6][BLOCK_PAIR];
    
    red_acc[0][tid]=f.x;
    red_acc[1][tid]=f.y;
    red_acc[2][tid]=f.z;
    red_acc[3][tid]=energy;

    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
      if (offset < s) {
        for (int r=0; r<4; r++)
          red_acc[r][tid] += red_acc[r][tid+s];
      }
    }
    
    f.x=red_acc[0][tid];
    f.y=red_acc[1][tid];
    f.z=red_acc[2][tid];
    energy=red_acc[3][tid];

    if (vflag>0) {
      for (int r=0; r<6; r++)
        red_acc[r][tid]=virial[r];

      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
        if (offset < s) {
          for (int r=0; r<6; r++)
            red_acc[r][tid] += red_acc[r][tid+s];
        }
      }
    
      for (int r=0; r<6; r++)
        virial[r]=red_acc[r][tid];
    }
  }

  // Store answers
  if (ii<inum && offset==0) {
    __global acctyp *ap1=engv+ii;
    if (eflag>0) {
      *ap1=energy;
      ap1+=inum;
    }
    if (vflag>0) {
      for (int i=0; i<6; i++) {
        *ap1=virial[i];
        ap1+=inum;
      }
    }
    ans[ii]=f;
  } // if ii
}

__kernel void kernel_pair_fast(__global numtyp4 *x_, __global numtyp4 *lj1_in,
                               __global numtyp4* lj3_in, 
                               __global numtyp* sp_lj_in, 
                               __global int *dev_nbor, __global int *dev_packed,
                               __global acctyp4 *ans, __global acctyp *engv, 
                               const int eflag, const int vflag, const int inum, 
                               const int nbor_pitch, const int t_per_atom) {
  int tid=THREAD_ID_X;
  int ii=mul24((int)BLOCK_ID_X,(int)(BLOCK_SIZE_X)/t_per_atom);
  ii+=tid/t_per_atom;
  int offset=tid%t_per_atom;

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    if (eflag>0)
      lj3[tid]=lj3_in[tid];
  }
  
  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(numtyp)0;
  
  __syncthreads();
  
  if (ii<inum) {
    __global int *nbor=dev_nbor+ii;
    int i=*nbor;
    nbor+=nbor_pitch;
    int numj=*nbor;
    nbor+=nbor_pitch;

    int n_stride;
    __global int *list_end;
    if (dev_nbor==dev_packed) {
      list_end=nbor+mul24(numj,nbor_pitch);
      nbor+=mul24(offset,nbor_pitch);
      n_stride=mul24(t_per_atom,nbor_pitch);
    } else {
      nbor=dev_packed+*nbor;
      list_end=nbor+numj;
      n_stride=t_per_atom;
      nbor+=offset;
    }
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    int iw=ix.w;
    int itype=mul24((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<list_end; nbor+=n_stride) {
  
      int j=*nbor;
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;
        
      if (r2inv<lj1[mtype].z) {
        numtyp r = sqrt(r2inv);
	numtyp rshift = r - lj1[mtype].w;
	numtyp rshiftsq = rshift*rshift;
	r2inv = 1.0/rshiftsq;
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        force*=factor_lj/rshift/r;
      
        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          energy+=factor_lj*(e-lj3[mtype].z); 
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  
  // Reduce answers
  if (t_per_atom>1) {
    __local acctyp red_acc[6][BLOCK_PAIR];
    
    red_acc[0][tid]=f.x;
    red_acc[1][tid]=f.y;
    red_acc[2][tid]=f.z;
    red_acc[3][tid]=energy;

    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
      if (offset < s) {
        for (int r=0; r<4; r++)
          red_acc[r][tid] += red_acc[r][tid+s];
      }
    }
    
    f.x=red_acc[0][tid];
    f.y=red_acc[1][tid];
    f.z=red_acc[2][tid];
    energy=red_acc[3][tid];

    if (vflag>0) {
      for (int r=0; r<6; r++)
        red_acc[r][tid]=virial[r];

      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
        if (offset < s) {
          for (int r=0; r<6; r++)
            red_acc[r][tid] += red_acc[r][tid+s];
        }
      }
    
      for (int r=0; r<6; r++)
        virial[r]=red_acc[r][tid];
    }
  }

  // Store answers
  if (ii<inum && offset==0) {
    __global acctyp *ap1=engv+ii;
    if (eflag>0) {
      *ap1=energy;
      ap1+=inum;
    }
    if (vflag>0) {
      for (int i=0; i<6; i++) {
        *ap1=virial[i];
        ap1+=inum;
      }
    }
    ans[ii]=f;
  } // if ii*/
}

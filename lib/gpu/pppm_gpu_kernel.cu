#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifndef PPPM_GPU_KERNEL
#define PPPM_GPU_KERNEL

#define OFFSET 16384

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp2 double2
#define numtyp4 double4
#define acctyp double
#define acctyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp double
#define acctyp4 double4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp float
#define acctyp4 float4
#endif

#ifdef NV_KERNEL

#include "geryon/ucl_nv_kernel.h"
texture<float4> pos_tex;
texture<float> q_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int& i, const double4 *pos)
{
  return pos[i];
}
__inline double fetch_q(const int& i, const double *q)
{
  return q[i];
}
#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(pos_tex, i);
}
__inline float fetch_q(const int& i, const float *q)
{
  return tex1Dfetch(q_tex, i);
}
#endif

#else

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#pragma OPENCL EXTENSION cl_khr_local_int32_base_atomics : enable
#define GLOBAL_ID_X get_global_id(0)
#define THREAD_ID_X get_local_id(0)
#define BLOCK_ID_X get_group_id(0)
#define BLOCK_SIZE_X get_local_size(0)
#define __syncthreads() barrier(CLK_LOCAL_MEM_FENCE)
#define __inline inline

#define fetch_pos(i,y) x_[i]
#define fetch_q(i,y) q_[i]

#endif

__kernel void particle_map(__global numtyp4 *x_, const int nlocal, 
                           __global int *counts, __global int *ans, 
                           const numtyp boxlo_x, const numtyp boxlo_y,
                           const numtyp boxlo_z, const numtyp delxinv,
                           const numtyp delyinv, const numtyp delzinv,
                           const int npts_x, const int npts_y,
                           const int npts_z, const int _brick_stride,
                           const int max_atoms, __global int *error) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;
  int nx,ny,nz;
  numtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);

    // Boxlo is adjusted to include ghost cells so that starting index is 0
    tx=(p.x-boxlo_x)*delxinv;
    nx=int(tx);
    ty=(p.y-boxlo_y)*delyinv;
    ny=int(ty);
    tz=(p.z-boxlo_z)*delzinv;
    nz=int(tz);

    if (tx<0 || ty<0 || tz<0 || nx>=npts_x || ny>=npts_y || nz>=npts_z)
      *error=1;
    else {
      int i=nz*npts_y*npts_x+ny*npts_x+nx;
      int old=atom_add(counts+i, 1);
      if (old==max_atoms)
        *error=2;
      else
        ans[_brick_stride*old+i]=ii;
    }
  }
}

/*
__kernel void particle_map(__global numtyp4 *x_, __global numtyp *q_,
                           __global int *counts, __global int *atoms, 
                           const numtyp boxlo_x, const numtyp boxlo_y,
                           const numtyp boxlo_z, const numtyp delxinv,
                           const numtyp delyinv, const numtyp delzinv,
                           const int npts_x, const int npts_y,
                           const int npts_z, const int _brick_stride,
                           const int max_atoms, __global int *error) {
  // ii indexes the two interacting particles in gi
  int xx=THREAD_ID_X;
  int yy=THREAD_ID_Y;
  int bx=BLOCK_ID_X;
  int by=BLOCK_ID_Y;
  int block_size=BLOCK_SIZE_X;
  
  int max_y=BLOCK_ID_Y*block_size+block_size;
  int max_x=BLOCK_ID_X*block_size+block_size;
  
  __local numtyp4 p;
  __local numtyp q,dx,dy,dz;
  __local int brick_i,count,atom_i;
  
  for (int z=-nlower; z<npts_z-order; z++)
    for (int ny=max_y-block_size; ny<max_y; ny++) {
      if (ny>npts_y)
        break;
      brick_i = z*npts_x*npts_y + ny*npts_x + max_x - block_size;
      for (int nx=max_x-block_size; nx<max_x; nx++) {
        if (nx>npts_x)
          break;
        count=counts[brick_i];
        for (int i=0; i<count; i++) {
          int atom_i=atoms[brick_i+i*_brick_stride];
          p=fetch_pos(x_,atom_i);
          q=fetch_q(q_,atom_i);
          
    dx = nx+shiftone - (x[i][0]-boxlo[0])*delxinv;
    dy = ny+shiftone - (x[i][1]-boxlo[1])*delyinv;
    dz = nz+shiftone - (x[i][2]-boxlo[2])*delzinv;
          
          
        
        
        
        brick_i++;
  
  
  int nx=GLOBAL_ID_X+nlower;
  int ny=GLOBAL_ID_Y+nlower;
  int block_size=BLOCK_SIZE_X;
  
    for (

  // (nx,ny,nz) = global coords of grid pt to "lower left" of charge
  // (dx,dy,dz) = distance to "lower left" grid pt
  // (mx,my,mz) = global coords of moving stencil pt

  double *q = atom->q;
  double **x = atom->x;
  int nlocal = atom->nlocal;

  for (int i = 0; i < nlocal; i++) {

    nx = part2grid[i][0];
    ny = part2grid[i][1];
    nz = part2grid[i][2];

    compute_rho1d(dx,dy,dz);

    z0 = delvolinv * q[i];
    for (n = nlower; n <= nupper; n++) {
      mz = n+nz;  // z-index of point being updated
      y0 = z0*rho1d[2][n]; 
      for (m = nlower; m <= nupper; m++) {
        my = m+ny;
        x0 = y0*rho1d[1][m];
        for (l = nlower; l <= nupper; l++) {
          mx = l+nx;
          density_brick[mz][my][mx] += x0*rho1d[0][l];
        }
      }
    }
  }
}

void PPPMGPU::compute_rho1d(double dx, double dy, double dz)
{
  int k,l;

  for (k = (1-order)/2; k <= order/2; k++) {
    rho1d[0][k] = 0.0;
    rho1d[1][k] = 0.0;
    rho1d[2][k] = 0.0;
    for (l = order-1; l >= 0; l--) {
      rho1d[0][k] = rho_coeff[l][k] + rho1d[0][k]*dx;
      rho1d[1][k] = rho_coeff[l][k] + rho1d[1][k]*dy;
      rho1d[2][k] = rho_coeff[l][k] + rho1d[2][k]*dz;
    }
  }
}

*/

#endif

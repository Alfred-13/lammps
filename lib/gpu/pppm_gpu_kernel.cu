#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifndef PPPM_GPU_KERNEL
#define PPPM_GPU_KERNEL

#define OFFSET 16384

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp2 double2
#define numtyp4 double4
#define acctyp double
#define acctyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp double
#define acctyp4 double4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp float
#define acctyp4 float4
#endif

#ifdef NV_KERNEL

#include "geryon/ucl_nv_kernel.h"
texture<float4> pos_tex;
texture<float> q_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int& i, const double4 *pos)
{
  return pos[i];
}
__inline double fetch_q(const int& i, const double *q)
{
  return q[i];
}
#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(pos_tex, i);
}
__inline float fetch_q(const int& i, const float *q)
{
  return tex1Dfetch(q_tex, i);
}
#endif

#else

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#pragma OPENCL EXTENSION cl_khr_local_int32_base_atomics : enable
#define GLOBAL_ID_X get_global_id(0)
#define THREAD_ID_X get_local_id(0)
#define BLOCK_ID_X get_group_id(0)
#define BLOCK_SIZE_X get_local_size(0)
#define __syncthreads() barrier(CLK_LOCAL_MEM_FENCE)
#define __inline inline

#define fetch_pos(i,y) x_[i]
#define fetch_q(i,y) q_[i]

#endif

__kernel void particle_map(__global numtyp4 *x_, const int nlocal, 
                           __global int *counts, __global int *ans, 
                           const numtyp boxlo_x, const numtyp boxlo_y,
                           const numtyp boxlo_z, const numtyp delxinv,
                           const numtyp delyinv, const numtyp delzinv,
                           const int npts_x, const int npts_y, const int npts_z,
                           const int max_atoms_m_1, __global int *error) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;
  int nx,ny,nz;
  numtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);

    // Boxlo is adjusted to include ghost cells so that starting index is 0
    tx=(p.x-boxlo_x)*delxinv;
    nx=int(tx);
    ty=(p.y-boxlo_y)*delyinv;
    ny=int(ty);
    tz=(p.z-boxlo_z)*delzinv;
    nz=int(tz);

    if (tx<0 || ty<0 || tz<0 || nx>=npts_x || ny>=npts_y || nz>=npts_z)
      *error=1;
    else {
      int old=atom_add(counts+nz*npts_y*npts_x+ny*npts_x+nx, 1);
      if (old==max_atoms_m_1) *error=2;
    }
  }
}

#endif


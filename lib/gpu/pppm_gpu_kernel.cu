#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifndef PPPM_GPU_KERNEL
#define PPPM_GPU_KERNEL

#define MAX_STENCIL 8
#define BLOCK_1D 64

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp2 double2
#define numtyp4 double4
#define acctyp double
#define acctyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp double
#define acctyp4 double4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp float
#define acctyp4 float4
#endif

#ifdef NV_KERNEL

#include "geryon/ucl_nv_kernel.h"
texture<float4> pos_tex;
texture<float> q_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int& i, const double4 *pos)
{
  return pos[i];
}
__inline double fetch_q(const int& i, const double *q)
{
  return q[i];
}

__device__ inline void atomicFloatAdd(double* address, double val) {
  double old = *address, assumed;
  do { 
    assumed = old;
    old = __longlong_as_double( atomicCAS((unsigned long long int*)address, 
                                          __double_as_longlong(assumed),
                                          __double_as_longlong(val +
                                          assumed)));
  } while (assumed != old); 
}

#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(pos_tex, i);
}
__inline float fetch_q(const int& i, const float *q)
{
  return tex1Dfetch(q_tex, i);
}

__device__ inline void atomicFloatAdd(float *address, float val)
{
       int i_val = __float_as_int(val);
       int tmp0 = 0;
       int tmp1;

       while( (tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0)
       {
               tmp0 = tmp1;
               i_val = __float_as_int(val + __int_as_float(tmp1));
       }
}


#endif

#else

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#pragma OPENCL EXTENSION cl_khr_local_int32_base_atomics : enable
#define GLOBAL_ID_X get_global_id(0)
#define THREAD_ID_X get_local_id(0)
#define BLOCK_ID_X get_group_id(0)
#define BLOCK_SIZE_X get_local_size(0)
#define __syncthreads() barrier(CLK_LOCAL_MEM_FENCE)
#define __inline inline

#define fetch_pos(i,y) x_[i]
#define fetch_q(i,y) q_[i]

#endif

__kernel void particle_map(__global numtyp4 *x_, const int nlocal, 
                           __global int *counts, __global int *ans, 
                           const numtyp b_lo_x, const numtyp b_lo_y,
                           const numtyp b_lo_z, const numtyp delxinv,
                           const numtyp delyinv, const numtyp delzinv,
                           const int nlocal_x, const int nlocal_y,
                           const int nlocal_z, const int atom_stride,
                           const int max_atoms, __global int *error) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;
  int nx,ny,nz;
  numtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);

    tx=(p.x-b_lo_x)*delxinv;
    nx=int(tx);
    ty=(p.y-b_lo_y)*delyinv;
    ny=int(ty);
    tz=(p.z-b_lo_z)*delzinv;
    nz=int(tz);

    if (tx<0 || ty<0 || tz<0 || nx>=nlocal_x || ny>=nlocal_y || nz>=nlocal_z)
      *error=1;
    else {
      int i=nz*nlocal_y*nlocal_x+ny*nlocal_x+nx;
      int old=atom_add(counts+i, 1);
      if (old==max_atoms) {
        *error=2;
        atom_add(counts+i,-1);
      } else
        ans[atom_stride*old+i]=ii;
    }
  }
}

__kernel void make_rho(__global numtyp4 *x_, __global numtyp *q_,
                       __global int *counts, __global int *atoms,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int atom_stride, const int npts_x,
                       const int npts_y, const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const int order,
                       const numtyp delvolinv) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];
  int nx=THREAD_ID_X;
  int ny=THREAD_ID_Y;
  if (nx<order && ny<order) {
    int ri=nx*order+ny;
    rho_coeff[ri]=_rho_coeff[ri];
  }
  __syncthreads();
  
  nx+=BLOCK_ID_X*BLOCK_SIZE_X;
  ny+=BLOCK_ID_Y*BLOCK_SIZE_Y;
  int nz=0;
  
  if (nx<nlocal_x && ny<nlocal_y) {
    int z_stride=nlocal_x*nlocal_y;
    int z_pos=nz*z_stride+ny*nlocal_x+nx;
    for ( ; nz<nlocal_z; nz++) {
      int natoms=counts[z_pos];
      for (int row=0; row<natoms; row++) {
        int atom=atoms[atom_stride*row+z_pos];
        numtyp4 p=fetch_pos(atom,x_);
        numtyp z0=delvolinv*fetch_q(atom,q_);
        
        numtyp dx = nx - (p.x-b_lo_x)*delxinv;
        numtyp dy = ny - (p.y-b_lo_y)*delyinv;
        numtyp dz = nz - (p.z-b_lo_z)*delzinv;

        numtyp rho1d[2][MAX_STENCIL];
        for (int k = 0; k < order; k++) {
          rho1d[0][k] = (numtyp)0.0;
          rho1d[1][k] = (numtyp)0.0;
          for (int l = order-1; l >= 0; l--) {
            rho1d[0][k] = rho_coeff[l*order+k] + rho1d[0][k]*dx;
            rho1d[1][k] = rho_coeff[l*order+k] + rho1d[1][k]*dy;
          }
        }
        
        for (int n = 0; n < order; n++) {
          numtyp rho1d_2 = (numtyp)0.0;
          for (int k = order-1; k >= 0; k--)
            rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
          numtyp y0 = z0*rho1d_2;
          int mz = (n+nz)*npts_y*npts_x + ny*npts_x +nx;
          for (int m = 0; m < order; m++) {
	          numtyp x0 = y0*rho1d[1][m];
	          for (int l = 0; l < order; l++) {
              atomicFloatAdd(brick+mz+l,x0*rho1d[0][l]);
	          }
	          mz+=npts_x;
	        }
	      }
	    }
	    z_pos+=z_stride;
	  }
	}
}

/* --------------------------- */

__kernel void make_rho2(__global numtyp4 *x_, __global numtyp *q_,
                       __global int *counts, __global int *atoms,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int atom_stride, const int npts_x,
                       const int npts_y, const int npts_z, 
                       const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const int nlow2,
                       const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const int order,
                       const numtyp delvolinv) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];
  __local numtyp front[BLOCK_1D+MAX_STENCIL];
  __local int nx,ny,x_start,y_start,x_stop,y_stop;
  __local int z_stride, z_local_stride;

  int tx=THREAD_ID_X;
  int tx_halo=BLOCK_1D+tx;
  if (tx<order*order)
    rho_coeff[tx]=_rho_coeff[tx];
    
  if (tx==0) {
    nx=BLOCK_ID_X;
    ny=BLOCK_ID_Y;
    x_start=0;
    y_start=0;
    x_stop=order;
    y_stop=order;
    if (nx<nlow2)
      x_start=nlow2-nx;
    if (ny<nlow2)
      y_start=nlow2-ny;
    if (nx>=nlocal_x)
      x_stop-=nx-nlocal_x+1;
    if (ny>=nlocal_y)
      y_stop-=ny-nlocal_y+1;
    z_stride=npts_x*npts_y*BLOCK_1D;
    z_local_stride=nlocal_x*nlocal_y*BLOCK_1D;
  }
  
  if (tx<order) 
    front[tx_halo]=(numtyp)0.0;
    
  __syncthreads();

  numtyp ans[MAX_STENCIL];
  int loop_count=npts_z/BLOCK_1D+1;
  int nz=tx;
  int pt = nz*npts_x*npts_y + ny*npts_x + nx;
  int z_local = nz*nlocal_x*nlocal_y;
  for (int i=0 ; i<loop_count; i++) {
    int offset1,offset2;
    if (i%2) {
      offset1=0;
      offset2=1;
    } else {
      offset1=1;
      offset2=0;
    }
    for (int n=0; n<order; n++)
      ans[n]=(numtyp)0.0;
    if (nz<nlocal_z) {
      for (int m=y_start; m<y_stop; m++) {
        int y_pos=ny+m-nlow2;
        int y_local=y_pos*nlocal_x;
        for (int l=x_start; l<x_stop; l++) {
          int x_pos=nx+l-nlow2;
          int pos=z_local+y_local+x_pos;
          int natoms=counts[pos]*atom_stride;
          for (int row=pos; row<natoms; row+=atom_stride) {
            int atom=atoms[row];
            numtyp4 p=fetch_pos(atom,x_);
            numtyp z0=delvolinv*fetch_q(atom,q_);
      
            numtyp dx = x_pos - (p.x-b_lo_x)*delxinv;
            numtyp dy = y_pos - (p.y-b_lo_y)*delyinv;
            numtyp dz = nz - (p.z-b_lo_z)*delzinv;
            
            numtyp rho1d_1 = (numtyp)0.0;
            numtyp rho1d_0 = (numtyp)0.0;
            for (int k = order; k > 0; k--) {
              rho1d_1 = rho_coeff[k*order-m-1] + rho1d_1*dy;
              rho1d_0 = rho_coeff[k*order-l-1] + rho1d_0*dx;
            }
            z0*=rho1d_1*rho1d_0;

            for (int n=0; n<order; n++) {
              numtyp rho1d_2 = (numtyp)0.0;
              for (int k = order-1; k >= 0; k--)
                rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
              ans[n]+=z0*rho1d_2;
            }
          }
        }
      }
    }
    
    __syncthreads();
    if (tx<order) {
      front[tx]=front[tx_halo];
      front[tx_halo]=(numtyp)0.0;
    } else 
      front[tx]=(numtyp)0.0;
    
    for (int n=0; n<order; n++) {
      front[tx+n]+=ans[n];
      __syncthreads();
    }

    if (nz<npts_z)
      brick[pt]=front[tx];
    nz+=BLOCK_1D;
    pt+=z_stride;
    z_local+=z_local_stride;
  }
}

/* --------------------------- */

__kernel void make_rho3(__global numtyp4 *x_, __global numtyp *q_,
                       const int nlocal,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int npts_x,
                       const int npts_y, const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const numtyp shift,
                       const int order,
                       const numtyp delvolinv, __global int *error) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];
  int ii=THREAD_ID_X;
  if (ii<order*order)
    rho_coeff[ii]=_rho_coeff[ii];
  __syncthreads();
  
  ii+=BLOCK_ID_X*BLOCK_SIZE_X;
  
  int nx,ny,nz;
  numtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);

    tx=(p.x-b_lo_x)*delxinv;
    nx=int(tx);
    ty=(p.y-b_lo_y)*delyinv;
    ny=int(ty);
    tz=(p.z-b_lo_z)*delzinv;
    nz=int(tz);

    if (tx<0 || ty<0 || tz<0 || nx>=nlocal_x || ny>=nlocal_y || nz>=nlocal_z)
      *error=1;
    else {
      numtyp z0=delvolinv*fetch_q(ii,q_);
        
      numtyp dx = nx+shift - tx;
      numtyp dy = ny+shift - ty;
      numtyp dz = nz+shift - tz;

      numtyp rho1d[2][MAX_STENCIL];
      for (int k = 0; k < order; k++) {
        rho1d[0][k] = (numtyp)0.0;
        rho1d[1][k] = (numtyp)0.0;
        for (int l = order-1; l >= 0; l--) {
          rho1d[0][k] = rho_coeff[l*order+k] + rho1d[0][k]*dx;
          rho1d[1][k] = rho_coeff[l*order+k] + rho1d[1][k]*dy;
        }
      }
        
      for (int n = 0; n < order; n++) {
        numtyp rho1d_2 = (numtyp)0.0;
        for (int k = order-1; k >= 0; k--)
          rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
        numtyp y0 = z0*rho1d_2;
        int mz = (n+nz)*npts_y*npts_x + ny*npts_x +nx;
        for (int m = 0; m < order; m++) {
          numtyp x0 = y0*rho1d[1][m];
	        for (int l = 0; l < order; l++) {
            atomicFloatAdd(brick+mz+l,x0*rho1d[0][l]);
	        }
          mz+=npts_x;
        }
	    }
	  }
	}
}

#endif


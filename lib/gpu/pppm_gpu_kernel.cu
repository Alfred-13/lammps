#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifndef PPPM_GPU_KERNEL
#define PPPM_GPU_KERNEL

#define OFFSET 16384

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp2 double2
#define numtyp4 double4
#define acctyp double
#define acctyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp double
#define acctyp4 double4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp float
#define acctyp4 float4
#endif

#ifdef NV_KERNEL

#include "geryon/ucl_nv_kernel.h"
texture<float4> pos_tex;
texture<float> q_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int& i, const double4 *pos)
{
  return pos[i];
}
__inline double fetch_q(const int& i, const double *q)
{
  return q[i];
}
#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(pos_tex, i);
}
__inline float fetch_q(const int& i, const float *q)
{
  return tex1Dfetch(q_tex, i);
}
#endif

#else

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#define GLOBAL_ID_X get_global_id(0)
#define THREAD_ID_X get_local_id(0)
#define BLOCK_ID_X get_group_id(0)
#define BLOCK_SIZE_X get_local_size(0)
#define __syncthreads() barrier(CLK_LOCAL_MEM_FENCE)
#define __inline inline

#define fetch_pos(i,y) x_[i]
#define fetch_q(i,y) q_[i]

#endif

__kernel void particle_map(__global numtyp4 *x_, const int nlocal, 
                           __global int *counts, __global int *ans, 
                           const numtyp boxlo_x, const numtyp boxlo_y,
                           const numtyp boxlo_z, const numtyp delxinv,
                           const numtyp delyinv, const numtyp delzinv,
                           const numtyp shift, const int nxlo_out,
                           const int nxhi_out, const int nylo_out,
                           const int nyhi_out, const int nzlo_out,
                           const int nzhi_out, const int nlower,
                           const int nupper, __global int *error) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;
  int nx,ny,nz;
/*
  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);
// shift boxlo

    nx = int((p.x-boxlo_x)*delxinv+shift) - OFFSET;
    ny = int((p.y-boxlo_y)*delyinv+shift) - OFFSET;
    nz = int((p.z-boxlo_z)*delzinv+shift) - OFFSET;
    counts[
    part2grid[i][0] = nx;
    part2grid[i][1] = ny;
    part2grid[i][2] = nz;

    // check that entire stencil around nx,ny,nz will fit in my 3d brick
    if (nx+nlower < nxlo_out || nx+nupper > nxhi_out ||
        ny+nlower < nylo_out || ny+nupper > nyhi_out ||
        nz+nlower < nzlo_out || nz+nupper > nzhi_out) *error=1;
  }*/
}

#endif


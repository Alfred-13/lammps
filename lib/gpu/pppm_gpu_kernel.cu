#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifndef PPPM_GPU_KERNEL
#define PPPM_GPU_KERNEL

#define MAX_STENCIL 8
#define BLOCK_1D 64

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp2 double2
#define numtyp4 double4
#define acctyp double
#define acctyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp double
#define acctyp4 double4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp float
#define acctyp4 float4
#endif

#ifdef NV_KERNEL

#include "geryon/ucl_nv_kernel.h"
texture<float4> pos_tex;
texture<float> q_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int& i, const double4 *pos)
{
  return pos[i];
}
__inline double fetch_q(const int& i, const double *q)
{
  return q[i];
}

__device__ inline void atomicFloatAdd(double* address, double val) {
  double old = *address, assumed;
  do { 
    assumed = old;
    old = __longlong_as_double( atomicCAS((unsigned long long int*)address, 
                                          __double_as_longlong(assumed),
                                          __double_as_longlong(val +
                                          assumed)));
  } while (assumed != old); 
}

#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(pos_tex, i);
}
__inline float fetch_q(const int& i, const float *q)
{
  return tex1Dfetch(q_tex, i);
}

__device__ inline void atomicFloatAdd(float *address, float val)
{
       int i_val = __float_as_int(val);
       int tmp0 = 0;
       int tmp1;

       while( (tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0)
       {
               tmp0 = tmp1;
               i_val = __float_as_int(val + __int_as_float(tmp1));
       }
}


#endif

#else

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#pragma OPENCL EXTENSION cl_khr_local_int32_base_atomics : enable
#define GLOBAL_ID_X get_global_id(0)
#define THREAD_ID_X get_local_id(0)
#define BLOCK_ID_X get_group_id(0)
#define BLOCK_SIZE_X get_local_size(0)
#define __syncthreads() barrier(CLK_LOCAL_MEM_FENCE)
#define __inline inline

#define fetch_pos(i,y) x_[i]
#define fetch_q(i,y) q_[i]

#endif

__kernel void particle_map(__global numtyp4 *x_, const int nlocal, 
                           __global int *counts, __global int *ans, 
                           const numtyp b_lo_x, const numtyp b_lo_y,
                           const numtyp b_lo_z, const numtyp delxinv,
                           const numtyp delyinv, const numtyp delzinv,
                           const int nlocal_x, const int nlocal_y,
                           const int nlocal_z, const int atom_stride,
                           const int max_atoms, __global int *error) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;
  int nx,ny,nz;
  numtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);

    tx=(p.x-b_lo_x)*delxinv;
    nx=int(tx);
    ty=(p.y-b_lo_y)*delyinv;
    ny=int(ty);
    tz=(p.z-b_lo_z)*delzinv;
    nz=int(tz);

    if (tx<0 || ty<0 || tz<0 || nx>=nlocal_x || ny>=nlocal_y || nz>=nlocal_z)
      *error=1;
    else {
      int i=nz*nlocal_y*nlocal_x+ny*nlocal_x+nx;
      int old=atom_add(counts+i, 1);
      if (old==max_atoms) {
        *error=2;
        atom_add(counts+i,-1);
      }
      else
        ans[atom_stride*old+i]=ii;
    }
  }
}

__kernel void make_rho(__global numtyp4 *x_, __global numtyp *q_,
                       __global int *counts, __global int *atoms,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int atom_stride, const int npts_x,
                       const int npts_y, const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const int order,
                       const numtyp delvolinv) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];
  int nx=THREAD_ID_X;
  int ny=THREAD_ID_Y;
  if (nx<order && ny<order) {
    int ri=nx*order+ny;
    rho_coeff[ri]=_rho_coeff[ri];
  }
  __syncthreads();
  
  nx+=BLOCK_ID_X*BLOCK_SIZE_X;
  ny+=BLOCK_ID_Y*BLOCK_SIZE_Y;
  int nz=0;
  
  if (nx<nlocal_x && ny<nlocal_y) {
    int z_stride=nlocal_x*nlocal_y;
    int z_pos=nz*z_stride+ny*nlocal_x+nx;
    for ( ; nz<nlocal_z; nz++) {
      int natoms=counts[z_pos];
      for (int row=0; row<natoms; row++) {
        int atom=atoms[atom_stride*row+z_pos];
        numtyp4 p=fetch_pos(atom,x_);
        numtyp z0=delvolinv*fetch_q(atom,q_);
        
        numtyp dx = nx - (p.x-b_lo_x)*delxinv;
        numtyp dy = ny - (p.y-b_lo_y)*delyinv;
        numtyp dz = nz - (p.z-b_lo_z)*delzinv;

        numtyp rho1d[2][MAX_STENCIL];
        for (int k = 0; k < order; k++) {
          rho1d[0][k] = (numtyp)0.0;
          rho1d[1][k] = (numtyp)0.0;
          for (int l = order-1; l >= 0; l--) {
            rho1d[0][k] = rho_coeff[l*order+k] + rho1d[0][k]*dx;
            rho1d[1][k] = rho_coeff[l*order+k] + rho1d[1][k]*dy;
          }
        }
        
        for (int n = 0; n < order; n++) {
          numtyp rho1d_2 = (numtyp)0.0;
          for (int k = order-1; k >= 0; k--)
            rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
          numtyp y0 = z0*rho1d_2;
          int mz = (n+nz)*npts_y*npts_x + ny*npts_x +nx;
          for (int m = 0; m < order; m++) {
	          numtyp x0 = y0*rho1d[1][m];
	          for (int l = 0; l < order; l++) {
              atomicFloatAdd(brick+mz+l,x0*rho1d[0][l]);
	          }
	          mz+=npts_x;
	        }
	      }
	    }
	    z_pos+=z_stride;
	  }
	}
}

/* --------------------------- */

__kernel void make_rho2(__global numtyp4 *x_, __global numtyp *q_,
                       __global int *counts, __global int *atoms,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int atom_stride, const int npts_x,
                       const int npts_y, const int npts_z, 
                       const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const int nlower,
                       const int nupper, const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const int order,
                       const numtyp delvolinv) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];
  __local int nx,ny,x_start,y_start,x_stop,y_stop,nlow2;

  int tx=THREAD_ID_X;
  if (tx==0) {
    nx=BLOCK_ID_X;
    ny=BLOCK_ID_Y;
    x_start=0;
    y_start=0;
    x_stop=order;
    y_stop=order;
    nlow2=nlower*-2;
    if (nx<nlow2)
      x_start=nlow2-nx;
    if (ny<nlow2)
      y_start=nlow2-ny;
    if (nx>=nlocal_x)
      x_stop-=nx-nlocal_x+1;
    if (ny>=nlocal_y)
      y_stop-=ny-nlocal_y+1;
  }
  
  if (tx<order*order)
    rho_coeff[tx]=_rho_coeff[tx];
  __syncthreads();

  numtyp ans[MAX_STENCIL];
  int loop_count=nlocal_z/BLOCK_1D+1;
  int nz=tx;
  for (int i=0 ; i<loop_count; i++) {
    for (int n=0; n<MAX_STENCIL; n++)
      ans[n]=(numtyp)0.0;
    if (nz<nlocal_z) {
      for (int m=y_start; m<y_stop; m++) {
        int y_pos=(ny+m-nlow2);
        for (int l=x_start; l<x_stop; l++) {
          int x_pos=nx+l-nlow2;
          int pos=nz*nlocal_x*nlocal_y+y_pos*nlocal_x+x_pos;
          int natoms=counts[pos];
          for (int row=0; row<natoms; row++) {
            int atom=atoms[atom_stride*row+pos];
            numtyp4 p=fetch_pos(atom,x_);
            numtyp z0=delvolinv*fetch_q(atom,q_);
      
            numtyp dx = x_pos - (p.x-b_lo_x)*delxinv;
            numtyp dy = y_pos - (p.y-b_lo_y)*delyinv;
            numtyp dz = nz - (p.z-b_lo_z)*delzinv;
            
            numtyp rho1d_1 = (numtyp)0.0;
            numtyp rho1d_0 = (numtyp)0.0;
            for (int k = order-1; k >= 0; k--) {
              rho1d_1 = rho_coeff[k*order+(order-m-1)] + rho1d_1*dy;
              rho1d_0 = rho_coeff[k*order+(order-l-1)] + rho1d_0*dx;
            }

            for (int n=0; n<order; n++) {
              numtyp rho1d_2 = (numtyp)0.0;
              for (int k = order-1; k >= 0; k--)
                rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
              numtyp y0 = z0*rho1d_2;
              numtyp x0 = y0*rho1d_1;
              ans[n]+=x0*rho1d_0;
            }
          }
        }
      }
    }
    for (int n=0; n<order; n++) {
      int pt = (nz+n)*npts_x*npts_y + ny*npts_x + nx;
      brick[pt]+=ans[n];
      __syncthreads();
    }
    nz+=BLOCK_1D;
  }
}

/* --------------------------- */

__kernel void make_rho3(__global numtyp4 *x_, __global numtyp *q_,
                       const int nlocal,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int npts_x,
                       const int npts_y, const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const numtyp shift,
                       const int order,
                       const numtyp delvolinv, __global int *error) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];
  int ii=THREAD_ID_X;
  if (ii<order*order)
    rho_coeff[ii]=_rho_coeff[ii];
  __syncthreads();
  
  ii+=BLOCK_ID_X*BLOCK_SIZE_X;
  
  int nx,ny,nz;
  numtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);

    tx=(p.x-b_lo_x)*delxinv;
    nx=int(tx);
    ty=(p.y-b_lo_y)*delyinv;
    ny=int(ty);
    tz=(p.z-b_lo_z)*delzinv;
    nz=int(tz);

    if (tx<0 || ty<0 || tz<0 || nx>=nlocal_x || ny>=nlocal_y || nz>=nlocal_z)
      *error=1;
    else {
      numtyp z0=delvolinv*fetch_q(ii,q_);
        
      numtyp dx = nx+shift - tx;
      numtyp dy = ny+shift - ty;
      numtyp dz = nz+shift - tz;

      numtyp rho1d[2][MAX_STENCIL];
      for (int k = 0; k < order; k++) {
        rho1d[0][k] = (numtyp)0.0;
        rho1d[1][k] = (numtyp)0.0;
        for (int l = order-1; l >= 0; l--) {
          rho1d[0][k] = rho_coeff[l*order+k] + rho1d[0][k]*dx;
          rho1d[1][k] = rho_coeff[l*order+k] + rho1d[1][k]*dy;
        }
      }
        
      for (int n = 0; n < order; n++) {
        numtyp rho1d_2 = (numtyp)0.0;
        for (int k = order-1; k >= 0; k--)
          rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
        numtyp y0 = z0*rho1d_2;
        int mz = (n+nz)*npts_y*npts_x + ny*npts_x +nx;
        for (int m = 0; m < order; m++) {
          numtyp x0 = y0*rho1d[1][m];
	        for (int l = 0; l < order; l++) {
            atomicFloatAdd(brick+mz+l,x0*rho1d[0][l]);
	        }
          mz+=npts_x;
        }
	    }
	  }
	}
}

#endif


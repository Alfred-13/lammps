#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifndef PPPM_GPU_KERNEL
#define PPPM_GPU_KERNEL

#define MAX_STENCIL 8
#define BLOCK_1D 64

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp2 double2
#define numtyp4 double4
#define acctyp double
#define acctyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp double
#define acctyp4 double4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp float
#define acctyp4 float4
#endif

#ifdef NV_KERNEL

#include "geryon/ucl_nv_kernel.h"
texture<float4> pos_tex;
texture<float> q_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int& i, const double4 *pos)
{
  return pos[i];
}
__inline double fetch_q(const int& i, const double *q)
{
  return q[i];
}

__device__ inline void atomicFloatAdd(double* address, double val) {
  double old = *address, assumed;
  do { 
    assumed = old;
    old = __longlong_as_double( atomicCAS((unsigned long long int*)address, 
                                          __double_as_longlong(assumed),
                                          __double_as_longlong(val +
                                          assumed)));
  } while (assumed != old); 
}

#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(pos_tex, i);
}
__inline float fetch_q(const int& i, const float *q)
{
  return tex1Dfetch(q_tex, i);
}

__device__ inline void atomicFloatAdd(float *address, float val)
{
       int i_val = __float_as_int(val);
       int tmp0 = 0;
       int tmp1;

       while( (tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0)
       {
               tmp0 = tmp1;
               i_val = __float_as_int(val + __int_as_float(tmp1));
       }
}


#endif

#else

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#pragma OPENCL EXTENSION cl_khr_local_int32_base_atomics : enable
#define GLOBAL_ID_X get_global_id(0)
#define THREAD_ID_X get_local_id(0)
#define BLOCK_ID_X get_group_id(0)
#define BLOCK_SIZE_X get_local_size(0)
#define __syncthreads() barrier(CLK_LOCAL_MEM_FENCE)
#define __inline inline

#define fetch_pos(i,y) x_[i]
#define fetch_q(i,y) q_[i]

#endif

__kernel void particle_map(__global numtyp4 *x_, const int nlocal, 
                           __global int *counts, __global int *ans, 
                           const numtyp b_lo_x, const numtyp b_lo_y,
                           const numtyp b_lo_z, const numtyp delxinv,
                           const numtyp delyinv, const numtyp delzinv,
                           const int nlocal_x, const int nlocal_y,
                           const int nlocal_z, const int atom_stride,
                           const int max_atoms, __global int *error) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;
  int nx,ny,nz;
  numtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);

    tx=(p.x-b_lo_x)*delxinv;
    nx=int(tx);
    ty=(p.y-b_lo_y)*delyinv;
    ny=int(ty);
    tz=(p.z-b_lo_z)*delzinv;
    nz=int(tz);

    if (tx<0 || ty<0 || tz<0 || nx>=nlocal_x || ny>=nlocal_y || nz>=nlocal_z)
      *error=1;
    else {
      int i=nz*nlocal_y*nlocal_x+ny*nlocal_x+nx;
      int old=atom_add(counts+i, 1);
      if (old==max_atoms) {
        *error=2;
        atom_add(counts+i,-1);
      }
      else
        ans[atom_stride*old+i]=ii;
    }
  }
}

__kernel void make_rho(__global numtyp4 *x_, __global numtyp *q_,
                       __global int *counts, __global int *atoms,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int atom_stride, const int npts_x,
                       const int npts_y, const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const int order,
                       const numtyp delvolinv) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];
  int nx=THREAD_ID_X;
  int ny=THREAD_ID_Y;
  if (nx<order && ny<order) {
    int ri=nx*order+ny;
    rho_coeff[ri]=_rho_coeff[ri];
  }
  __syncthreads();
  
  nx+=BLOCK_ID_X*BLOCK_SIZE_X;
  ny+=BLOCK_ID_Y*BLOCK_SIZE_Y;
  int nz=0;
  
  if (nx<nlocal_x && ny<nlocal_y) {
    int z_stride=nlocal_x*nlocal_y;
    int z_pos=nz*z_stride+ny*nlocal_x+nx;
    for ( ; nz<nlocal_z; nz++) {
      int natoms=counts[z_pos];
      for (int row=0; row<natoms; row++) {
        int atom=atoms[atom_stride*row+z_pos];
        numtyp4 p=fetch_pos(atom,x_);
        numtyp z0=delvolinv*fetch_q(atom,q_);
        
        numtyp dx = nx - (p.x-b_lo_x)*delxinv;
        numtyp dy = ny - (p.y-b_lo_y)*delyinv;
        numtyp dz = nz - (p.z-b_lo_z)*delzinv;

        numtyp rho1d[2][MAX_STENCIL];
        for (int k = 0; k < order; k++) {
          rho1d[0][k] = (numtyp)0.0;
          rho1d[1][k] = (numtyp)0.0;
          for (int l = order-1; l >= 0; l--) {
            rho1d[0][k] = rho_coeff[l*order+k] + rho1d[0][k]*dx;
            rho1d[1][k] = rho_coeff[l*order+k] + rho1d[1][k]*dy;
          }
        }
        
        for (int n = 0; n < order; n++) {
          numtyp rho1d_2 = (numtyp)0.0;
          for (int k = order-1; k >= 0; k--)
            rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
          numtyp y0 = z0*rho1d_2;
          int mz = (n+nz)*npts_y*npts_x + ny*npts_x +nx;
          for (int m = 0; m < order; m++) {
	          numtyp x0 = y0*rho1d[1][m];
	          for (int l = 0; l < order; l++) {
              atomicFloatAdd(brick+mz+l,x0*rho1d[0][l]);
	          }
	          mz+=npts_x;
	        }
	      }
	    }
	    z_pos+=z_stride;
	  }
	}
}

/* --------------------------- */

__kernel void make_rho2(__global numtyp4 *x_, __global numtyp *q_,
                       __global int *counts, __global int *atoms,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int atom_stride, const int npts_x,
                       const int npts_y, const int npts_z, 
                       const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const int nlower,
                       const int nupper, const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const int order,
                       const numtyp delvolinv) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];

  int nx=BLOCK_ID_X;
  int ny=BLOCK_ID_Y;
  int tx=THREAD_ID_X;
  if (tx<order*order)
    rho_coeff[tx]=_rho_coeff[tx];
  __syncthreads();

  if (nx<npts_x && ny<npts_y) {
    int x_start=0;
    int y_start=0;
    int x_stop=order;
    int y_stop=order;
    if (nx<-nlower)
      x_start=-(nx+nlower);
    if (ny<-nlower)
      y_start=-(ny+nlower);
    if (nx>=nlocal_x)
      x_stop-=nx-nlocal_x+1;
    if (ny>=nlocal_y)
      y_stop-=ny-nlocal_y+1;

    for (int nz=tx ; nz<npts_z; nz+=BLOCK_1D) {
      int pt = nz*npts_x*npts_y + ny*npts_x + nx;

      int z_start=0;
      int z_stop=order;
      if (nz<-nlower)
        z_start=-(nz+nlower);
      if (nz>=nlocal_z)
        z_stop-=nz-nlocal_z+1;

      for (int n=z_start; n<z_stop; n++) {
        int z_pos=(nz+n+nlower);
        for (int m=y_start; m<y_stop; m++) {
          int y_pos=(ny+m+nlower);
          for (int l=x_start; l<x_stop; l++) {
            int x_pos=nx+l+nlower;
            int pos=z_pos*nlocal_x*nlocal_y+y_pos*nlocal_x+x_pos;
            int natoms=counts[pos];
            for (int row=0; row<natoms; row++) {
              int atom=atoms[atom_stride*row+pos];
              numtyp4 p=fetch_pos(atom,x_);
              numtyp z0=delvolinv*fetch_q(atom,q_);
        
              numtyp dx = x_pos - (p.x-b_lo_x)*delxinv;
              numtyp dy = y_pos - (p.y-b_lo_y)*delyinv;
              numtyp dz = z_pos - (p.z-b_lo_z)*delzinv;
            
              numtyp rho1d_2 = (numtyp)0.0;
              numtyp rho1d_1 = (numtyp)0.0;
              numtyp rho1d_0 = (numtyp)0.0;
              for (int k = order-1; k >= 0; k--) {
                rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
                rho1d_1 = rho_coeff[k*order+m] + rho1d_1*dy;
                rho1d_0 = rho_coeff[k*order+l] + rho1d_0*dx;
              }
            
              numtyp y0 = z0*rho1d_2;
    	        numtyp x0 = y0*rho1d_1;
              brick[pt]=p.x;
  	        }
  	      }
  	    }
	    }
	  }
	}
}

/* --------------------------- */

__kernel void make_rho3(__global numtyp4 *x_, __global numtyp *q_,
                       const int nlocal,
                       __global numtyp *brick, __global numtyp *_rho_coeff,
                       const int npts_x,
                       const int npts_y, const int nlocal_x, const int nlocal_y,
                       const int nlocal_z, const numtyp b_lo_x,
                       const numtyp b_lo_y, const numtyp b_lo_z,
                       const numtyp delxinv, const numtyp delyinv,
                       const numtyp delzinv, const numtyp shift,
                       const int order,
                       const numtyp delvolinv, __global int *error) {
  __local numtyp rho_coeff[MAX_STENCIL*MAX_STENCIL];
  int ii=THREAD_ID_X;
  if (ii<order*order)
    rho_coeff[ii]=_rho_coeff[ii];
  __syncthreads();
  
  ii+=BLOCK_ID_X*BLOCK_SIZE_X;
  
  int nx,ny,nz;
  numtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);

    tx=(p.x-b_lo_x)*delxinv;
    nx=int(tx);
    ty=(p.y-b_lo_y)*delyinv;
    ny=int(ty);
    tz=(p.z-b_lo_z)*delzinv;
    nz=int(tz);

    if (tx<0 || ty<0 || tz<0 || nx>=nlocal_x || ny>=nlocal_y || nz>=nlocal_z)
      *error=1;
    else {
      numtyp z0=delvolinv*fetch_q(ii,q_);
        
      numtyp dx = nx+shift - tx;
      numtyp dy = ny+shift - ty;
      numtyp dz = nz+shift - tz;

      numtyp rho1d[2][MAX_STENCIL];
      for (int k = 0; k < order; k++) {
        rho1d[0][k] = (numtyp)0.0;
        rho1d[1][k] = (numtyp)0.0;
        for (int l = order-1; l >= 0; l--) {
          rho1d[0][k] = rho_coeff[l*order+k] + rho1d[0][k]*dx;
          rho1d[1][k] = rho_coeff[l*order+k] + rho1d[1][k]*dy;
        }
      }
        
      for (int n = 0; n < order; n++) {
        numtyp rho1d_2 = (numtyp)0.0;
        for (int k = order-1; k >= 0; k--)
          rho1d_2 = rho_coeff[k*order+n] + rho1d_2*dz;
        numtyp y0 = z0*rho1d_2;
        int mz = (n+nz)*npts_y*npts_x + ny*npts_x +nx;
        for (int m = 0; m < order; m++) {
          numtyp x0 = y0*rho1d[1][m];
	        for (int l = 0; l < order; l++) {
            atomicFloatAdd(brick+mz+l,x0*rho1d[0][l]);
	        }
          mz+=npts_x;
        }
	    }
	  }
	}
}

#endif


#include "hip/hip_runtime.h"
// **************************************************************************
//                               cg_cmm_msm.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for acceleration of the cg/cmm/msm pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : 
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
texture<float4> pos_tex;
texture<float> q_tex;
#ifndef _DOUBLE_DOUBLE
ucl_inline float4 fetch_pos(const int& i, const float4 *pos) 
  { return tex1Dfetch(pos_tex, i); }
ucl_inline float fetch_q(const int& i, const float *q) 
  { return tex1Dfetch(q_tex, i); }
#endif
#endif

__kernel void kernel_pair(__global numtyp4 *x_, __global numtyp4 *lj1,
                          __global numtyp4* lj3, const int lj_types, 
                          __global numtyp *sp_lj_in, __global int *dev_nbor, 
                          __global int *dev_packed, __global acctyp4 *ans,
                          __global acctyp *engv, const int eflag,
                          const int vflag, const int inum,
                          const int nbor_pitch, __global numtyp *q_,
                          const numtyp cut_coulsq, const numtyp qqrd2e,
                          const int smooth, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[8];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];
  sp_lj[4]=sp_lj_in[4];
  sp_lj[5]=sp_lj_in[5];
  sp_lj[6]=sp_lj_in[6];
  sp_lj[7]=sp_lj_in[7];
  __local numtyp _ia;
  __local numtyp _ia2;
  __local numtyp _ia3;

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  
  if (ii<inum) {
    _ia=(numtyp)-1.0/sqrt(cut_coulsq);
    _ia2=(numtyp)-1.0/cut_coulsq;
    _ia3=_ia2*_ia;
    
    __global int *nbor, *list_end;
    int i, numj, n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    numtyp qtmp=fetch_q(i,q_);
    int itype=ix.w;

    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = (numtyp)1.0-sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<lj1[mtype].x) {
        numtyp forcecoul, force_lj, force, inv1, inv2, prefactor;
        numtyp r2inv=(numtyp)1.0/rsq;

        if (rsq < lj1[mtype].y) {
          if (lj3[mtype].x == (numtyp)2) {
            inv1=r2inv*r2inv;
            inv2=inv1*inv1;
          } else if (lj3[mtype].x == (numtyp)1) {
            inv2=r2inv*sqrt(r2inv);
            inv1=inv2*inv2;
          } else {
            inv1=r2inv*r2inv*r2inv;
            inv2=inv1;
          }
          force_lj = factor_lj*inv1*(lj1[mtype].z*inv2-lj1[mtype].w);
        } else
          force_lj = (numtyp)0.0;

        numtyp ir, r2_ia2, r4_ia4, r6_ia6;
        if (rsq < cut_coulsq) {
          ir = (numtyp)1.0/sqrt(rsq);
          prefactor = qqrd2e*qtmp*fetch_q(j,q_);
          r2_ia2 = rsq*_ia2;
          r4_ia4 = r2_ia2*r2_ia2;
          if (smooth==0)
            forcecoul = prefactor*(_ia3*((numtyp)-4.375+(numtyp)5.25*r2_ia2-
                                        (numtyp)1.875*r4_ia4)-ir/rsq-
                                        factor_coul*ir);
          else {
            r6_ia6 = r2_ia2*r4_ia4;
            forcecoul = prefactor*(_ia3*((numtyp)-6.5625+(numtyp)11.8125*
                                         r2_ia2-(numtyp)8.4375*r4_ia4+
                                         (numtyp)2.1875*r6_ia6)-ir/rsq-
                                         factor_coul*ir);
          }
        } else
          forcecoul = (numtyp)0.0;

        force = forcecoul + force_lj * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          if (rsq < cut_coulsq)
            if (smooth==0)
              e_coul += prefactor*(ir+_ia*((numtyp)2.1875-(numtyp)2.1875*r2_ia2+
                                           (numtyp)1.3125*r4_ia4-
                                           (numtyp)0.3125*r4_ia4*r2_ia2)-
                                           factor_coul*ir);
            else
              e_coul += prefactor*(ir+_ia*((numtyp)2.4609375-(numtyp)3.28125*
                                           r2_ia2+(numtyp)2.953125*r4_ia4-
                                           (numtyp)1.40625*r6_ia6+
                                           (numtyp)0.2734375*r4_ia4*r4_ia4));
              
          if (rsq < lj1[mtype].y) {
            energy += factor_lj*inv1*(lj3[mtype].y*inv2-lj3[mtype].z)-
                      lj3[mtype].w;
          } 
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

__kernel void kernel_pair_fast(__global numtyp4 *x_, __global numtyp4 *lj1_in,
                               __global numtyp4* lj3_in, 
                               __global numtyp* sp_lj_in,
                               __global int *dev_nbor, __global int *dev_packed,
                               __global acctyp4 *ans, __global acctyp *engv, 
                               const int eflag, const int vflag, const int inum, 
                               const int nbor_pitch, __global numtyp *q_,
                               const numtyp cut_coulsq, const numtyp qqrd2e,
                               const int smooth, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[8];
  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    lj3[tid]=lj3_in[tid];
  }
  
  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  
  __local numtyp _ia;
  __local numtyp _ia2;
  __local numtyp _ia3;
  _ia=(numtyp)-1.0/sqrt(cut_coulsq);
  _ia2=(numtyp)1.0/cut_coulsq;
  _ia3=_ia2*_ia;
  __syncthreads();
  
  if (ii<inum) {
    __global int *nbor, *list_end;
    int i, numj, n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    numtyp qtmp=fetch_q(i,q_);
    int iw=ix.w;
    int itype=mul24((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = (numtyp)1.0-sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<lj1[mtype].x) {
        numtyp forcecoul, force_lj, force, inv1, inv2, prefactor;
        numtyp r2inv=(numtyp)1.0/rsq;

        if (rsq < lj1[mtype].y) {
          if (lj3[mtype].x == (numtyp)2) {
            inv1=r2inv*r2inv;
            inv2=inv1*inv1;
          } else if (lj3[mtype].x == (numtyp)1) {
            inv2=r2inv*sqrt(r2inv);
            inv1=inv2*inv2;
          } else {
            inv1=r2inv*r2inv*r2inv;
            inv2=inv1;
          }
          force_lj = factor_lj*inv1*(lj1[mtype].z*inv2-lj1[mtype].w);
        } else
          force_lj = (numtyp)0.0;

        numtyp ir, r2_ia2, r4_ia4, r6_ia6;
        if (rsq < cut_coulsq) {
          ir = (numtyp)1.0/sqrt(rsq);
          prefactor = qqrd2e*qtmp*fetch_q(j,q_);
          r2_ia2 = rsq*_ia2;
          r4_ia4 = r2_ia2*r2_ia2;
          if (smooth==0)
            forcecoul = prefactor*(_ia3*((numtyp)-4.375+(numtyp)5.25*r2_ia2-
                                        (numtyp)1.875*r4_ia4)-ir/rsq-
                                        factor_coul*ir);
          else {
            r6_ia6 = r2_ia2*r4_ia4;
            forcecoul = prefactor*(_ia3*((numtyp)-6.5625+(numtyp)11.8125*
                                         r2_ia2-(numtyp)8.4375*r4_ia4+
                                         (numtyp)2.1875*r6_ia6)-ir/rsq-
                                         factor_coul*ir);
          }
        } else
          forcecoul = (numtyp)0.0;

        force = forcecoul + force_lj * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          if (rsq < cut_coulsq)
            if (smooth==0)
              e_coul += prefactor*(ir+_ia*((numtyp)2.1875-(numtyp)2.1875*r2_ia2+
                                           (numtyp)1.3125*r4_ia4-
                                           (numtyp)0.3125*r4_ia4*r2_ia2)-
                                           factor_coul*ir);
            else
              e_coul += prefactor*(ir+_ia*((numtyp)2.4609375-(numtyp)3.28125*
                                           r2_ia2+(numtyp)2.953125*r4_ia4-
                                           (numtyp)1.40625*r6_ia6+
                                           (numtyp)0.2734375*r4_ia4*r4_ia4));
          if (rsq < lj1[mtype].y) {
            energy += factor_lj*inv1*(lj3[mtype].y*inv2-lj3[mtype].z)-
                      lj3[mtype].w;
          } 
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

